#include "hip/hip_runtime.h"
#define STB_IMAGE_IMPLEMENTATION
#define STB_IMAGE_WRITE_IMPLEMENTATION

#include <iostream>
#include <ctime>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <climits>
#include <string>
#include <math.h>
#include <time.h>
#include <chrono> 
#include "define.h"
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <hiprand.h>

using namespace std;

#include "stb_image.h"
#include "stb_image_write.h"

#define ENABLE_TIMER 1

__device__ __host__ int MAX(int a, int b) {
  if (a > b){
    return a;
  }
  return b;
}
__device__ __host__ int MIN(int a, int b) {
  if (a < b){
    return a;
  }
  return b;
}

__device__ __host__ unsigned char* get_RGB(unsigned char* img_pixels, int img_no, int row, int col, int nchannels, int h, int w){
  unsigned bytePerPixel = nchannels;
  unsigned char* pixelOffset = img_pixels + (h*w*img_no + (row*w + col)) * bytePerPixel;
  return pixelOffset;
}

__device__ __host__ void storeXYasRGB(unsigned char* offset, int bx, int by, int bz)
{
    offset[0] = bx;
    offset[1] = by;
    offset[2] = bz;
}


__device__ __host__ void storeintasRGBA(unsigned char* offset, int tostore)
{
    offset[3] = tostore&0xff;
    offset[2] = (tostore&0xff00)>>8;
    offset[1] = (tostore&0xff0000)>>16;
    offset[0] = (tostore&0xff000000)>>24;
}

__device__ __host__ int dist1(unsigned char *a, unsigned char *b, int ax, int ay, int az, int bx, int by, int bz, int pw, int h, int w, int cutoff=INT_MAX) {
  // printf("PW %d\n", pw);
  int ans = 0;
  for (int dz = 0; dz < pw; dz ++) {
    for (int dy = 0; dy < pw; dy++) {
      for (int dx = 0; dx < pw; dx++) {
        unsigned char* ac = get_RGB(a, az + dz, ay+dy, ax+dx, 3, h, w);
        unsigned char* bc = get_RGB(b, bz + dz, by+dy, bx+dx, 3, h, w);
        int dr = ac[0] - bc[0];
        int dg = ac[1] - bc[1];
        int db = ac[2] - bc[2];
        ans += dr*dr + dg*dg + db*db;
      }
    }
    if (ans >= cutoff) { return cutoff; }
    if (ans < 0){ return cutoff; }
  }
  // printf("dist being entered\n");

  return ans;
}



__device__ __host__ int dist(unsigned char *a, unsigned char *b, int ax, int ay, int az, int bx, int by, int bz, int pw, int h, int w, int cutoff=INT_MAX) {
  // printf("PW %d\n", pw);
  int ans = 0;
  for (int dz = 0; dz < pw; dz ++) {
    for (int dy = 0; dy < pw; dy++) {
      for (int dx = 0; dx < pw; dx++) {
        unsigned char* ac = get_RGB(a, az + dz, ay+dy, ax+dx, 3, h, w);
        unsigned char* bc = get_RGB(b, bz + dz, by+dy, bx+dx, 3, h, w);
        int dr = ac[0] - bc[0];
        int dg = ac[1] - bc[1];
        int db = ac[2] - bc[2];
        ans += dr*dr + dg*dg + db*db;
      }
    }
    if (ans >= cutoff) { return cutoff; }
    if (ans < 0){ return cutoff; }
  }
  // printf("dist being entered\n");

  return ans;
}


__device__ __host__ void improve_guess(unsigned char *a, unsigned char *b, int ax, int ay, int az, int &xbest, int &ybest, int &zbest, int &dbest, int bx, int by, int bz, int pw, int h, int w) {
  // printf("PW %d\n", pw);
  
  int d = dist1(a, b, ax, ay, az, bx, by, bz, pw, h, w, dbest);
  // printf("dist is updating\n");
  if (d < dbest) {
    dbest = d;
    xbest = bx;
    ybest = by;
    zbest = bz;
  }
}

__device__  __host__ int XfromRGB(unsigned char* v){
    return v[0];
}

__device__  __host__ int YfromRGB(unsigned char* v){
    return v[1];
}

__device__  __host__ int ZfromRGB(unsigned char* v){
    return v[2];
}

__device__  __host__ int netINT(unsigned char *v){
    return v[0]<<24 | v[1] << 16 | v[2] << 8 | v[3];
}


void map_patches(unsigned char *a, unsigned char *b, unsigned char *ann, unsigned char *out)
{
  int aew = w - patch_width + 1, aeh = h - patch_width + 1, aed = d - patch_width + 1;
  for (int az = 0; az < aed; az ++) {
    for (int ay = 0; ay < aeh; ay++) {
      for (int ax = 0; ax < aew; ax++) {
        unsigned char* map = get_RGB(ann, az, ay, ax, 3, h, w);
        int x = XfromRGB(map);
        int y = YfromRGB(map);
        int z = ZfromRGB(map);
        for (int dz = 0; dz < patch_width; dz ++) {
          for (int dy = 0; dy < patch_width; dy++) {
            for (int dx = 0; dx < patch_width; dx++) {
              if (z+dz >= d || y+dy >= h || x+dx >= w || az + dz >=  d || ay + dy >=  h || ax + dx >= w)
                continue;
              unsigned char* binputs = get_RGB(b, z + dz, y+dy, x+dx, 3, h, w);
              unsigned char* output = get_RGB(out, az + dz, ay+dy, ax+dx, 3, h, w);
              output[0] = binputs[0];
              output[1] = binputs[1];
              output[2] = binputs[2];
            }
          }
        }
      }
    }
  }
}


__global__ void patchmatch(unsigned char *a, unsigned char *b, unsigned char *ann_to_use, unsigned char *annd_to_use, unsigned char *other, unsigned char *annd_other, int height, int width, int depth, int pw)
{
  // printf("LAUNCHED\n");
  int ax = threadIdx.x + blockDim.x*blockIdx.x;
  int ay = threadIdx.y + blockDim.y*blockIdx.y;
  int az = threadIdx.z + blockDim.z*blockIdx.z;
  int xchange = 1;
  int ychange = 1;
  int zchange = 1;
  int aew = width - pw + 1, aeh = height - pw + 1, aed = depth - pw + 1;
  int bew = width - pw + 1, beh = height - pw + 1, bed = depth - pw + 1;

  if (ax < aew && ay < aeh && az < aed)
  {
  unsigned char* v = get_RGB(ann_to_use, az, ay, ax, 3, height, width);
  int ybest = YfromRGB(v), xbest = XfromRGB(v), zbest = ZfromRGB(v);
  int dbest = netINT(get_RGB(annd_to_use, az, ay, ax,4, height, width));



  // Propagation
  if ((unsigned) (ax - xchange) < (unsigned) aew) {
    unsigned char* vp = get_RGB(ann_to_use, az, ay, ax-xchange, 3, height, width);
    int xp = XfromRGB(vp) + xchange, yp = YfromRGB(vp), zp = ZfromRGB(vp);
    if ((unsigned) xp < (unsigned) bew) {
      improve_guess(a, b, ax, ay, az, xbest, ybest, zbest, dbest, xp, yp, zp, pw, height, width);      
    }
    
  }

  if ((unsigned) (ay - ychange) < (unsigned) aeh) {
    unsigned char* vp = get_RGB(ann_to_use, az, ay-ychange, ax, 3, height, width);
    int xp = XfromRGB(vp), yp = YfromRGB(vp) + ychange, zp = ZfromRGB(vp);
    if ((unsigned) yp < (unsigned) beh) {
      improve_guess(a, b, ax, ay, az, xbest, ybest, zbest, dbest, xp, yp, zp, pw, height, width);
    }
  }

  if ((unsigned) (az - zchange) < (unsigned) aed) {
    unsigned char* vp = get_RGB(ann_to_use, az-zchange, ay, ax, 3, height, width);
    int xp = XfromRGB(vp), yp = YfromRGB(vp), zp = ZfromRGB(vp) + zchange;
    if ((unsigned) zp < (unsigned) bed) {
      improve_guess(a, b, ax, ay, az, xbest, ybest, zbest, dbest, xp, yp, zp, pw, height, width);
    }
  }

  if ((unsigned) (ax + xchange) < (unsigned) aew) {
    unsigned char* vp = get_RGB(ann_to_use, az, ay, ax + xchange, 3, height, width);
    int xp = XfromRGB(vp) - xchange, yp = YfromRGB(vp), zp = ZfromRGB(vp);
    if ((unsigned) xp < (unsigned) bew) {
      improve_guess(a, b, ax, ay, az, xbest, ybest, zbest, dbest, xp, yp, zp, pw, height, width);
    }
  }

  if ((unsigned) (ay + ychange) < (unsigned) aeh) {
    unsigned char* vp = get_RGB(ann_to_use,az, ay + ychange, ax, 3, height, width);
    int xp = XfromRGB(vp), yp = YfromRGB(vp) - ychange, zp = ZfromRGB(vp);
    if ((unsigned) yp < (unsigned) beh) {
      improve_guess(a, b, ax, ay, az, xbest, ybest, zbest, dbest, xp, yp, zp, pw, height, width);
    }
  }

  if ((unsigned) (az + zchange) < (unsigned) aed) {
    unsigned char* vp = get_RGB(ann_to_use, az + zchange, ay, ax, 3, height, width);
    int xp = XfromRGB(vp), yp = YfromRGB(vp), zp = ZfromRGB(vp) - zchange;
    if ((unsigned) zp < (unsigned) bed) {
      improve_guess(a, b, ax, ay, az, xbest, ybest, zbest, dbest, xp, yp, zp, pw, height, width);
    }
  }

  // Propagation done!

  int rs_start = INT_MAX;
  if (rs_start > MAX(MAX(width, height), depth)) { rs_start = MAX(MAX(width, height), depth); }
  for (int mag = rs_start; mag >= 1; mag /= 2) {
    int xmin = MAX(xbest-mag, 0), xmax = MIN(xbest+mag+1,bew);
    int ymin = MAX(ybest-mag, 0), ymax = MIN(ybest+mag+1,beh);
    int zmin = MAX(zbest-mag, 0), zmax = MIN(zbest+mag+1,bed);

    int xp = xmin;
    int yp = ymin;
    int zp = zmin;

    if (xmax > xmin)
    {
      xp = xmin;
    }
    if (ymax > ymin)
    {
      yp = ymin;
    }
    if (zmax > zmin)
    {
      zp = zmin;
    }
    improve_guess(a, b, ax, ay, az, xbest, ybest, zbest, dbest, xp, yp, zp, pw, height, width);
  }

  storeXYasRGB(get_RGB(other, az, ay, ax, 3, height, width), xbest, ybest, zbest);
  storeintasRGBA(get_RGB(annd_other, az, ay, ax, 4, height, width), dbest);
}
}

double compute_error(unsigned char* output, unsigned char* a)
{
  double error = 0;

  for (int az = 0; az < d; az++) {
    for (int ay = 0; ay < h; ay++) {
      for (int ax = 0; ax < w; ax++){
        unsigned char* pixelOffset = get_RGB(output, az, ay, ax, 3, h, w);
        unsigned char* pixelOffset_a = get_RGB(a, az, ay, ax, 3, h, w);
        int r = pixelOffset[0] - pixelOffset_a[0];
        int g = pixelOffset[1] - pixelOffset_a[1];
        int b = pixelOffset[2] - pixelOffset_a[2];

        double pixelError = r*r + g*g + b*b;

        error += sqrt(pixelError);
      }
    }
  }

  return error;
}

int main(int argc, char **argv) {

  // current assumption is that image sizes are same, update it later

  int height = h;
  int width = w;
  int depth = d;
  int nchannels = 3;
  unsigned char* img1_pixels;
  unsigned char* img2_pixels;


  
  if(argc < 3) {
    cout<<"Usage: " << argv[0] << " <image_file1>  <image_file2>\n";
    return 1;
    }
  // cout << w << h << d << endl;
  // Read input image 1
    img1_pixels = (unsigned char*) malloc(sizeof(unsigned char)*width*height*depth*nchannels);
  for (int i = 0; i < d; i ++){
    int temp_width, temp_height, temp_nchannels;
    string path= argv[1]+ to_string(i) +".jpeg";
    unsigned char *img_in = stbi_load(path.c_str(), &temp_width, &temp_height, &temp_nchannels, 0);
    int start_idx = width * height * i * nchannels;
    memcpy((void*)(img1_pixels + start_idx), img_in, sizeof(unsigned char)*temp_width*temp_height*temp_nchannels);
  }

  // Read input image 2
    img2_pixels = (unsigned char*) malloc(sizeof(unsigned char)*width*height*depth*nchannels);
  for (int i = 0; i < d; i ++){
    int temp_width, temp_height, temp_nchannels;
    string path= argv[2]+ to_string(i) +".jpeg";
    unsigned char *img_in = stbi_load(path.c_str(), &temp_width, &temp_height, &temp_nchannels, 0);
    int start_idx = width * height * i * nchannels;
    memcpy((void*)(img2_pixels + start_idx), img_in, sizeof(unsigned char)*temp_width*temp_height*temp_nchannels);
  }

  
  unsigned char* ann = (unsigned char*) malloc(sizeof(unsigned char)*width*height*depth*3);
  unsigned char* ann_buf = (unsigned char*) malloc(sizeof(unsigned char)*width*height*depth*3);
  unsigned char* annd = (unsigned char*) malloc(sizeof(unsigned char)*width*height*depth*4);
  unsigned char* annd_buf = (unsigned char*) malloc(sizeof(unsigned char)*width*height*depth*4);


  unsigned char* d_img1_pixels;
  unsigned char* d_img2_pixels;
  unsigned char* d_ann;
  unsigned char* d_annd;
  unsigned char* d_ann_buf;
  unsigned char* d_annd_buf;



  hipMalloc((void**)&d_img1_pixels, sizeof(unsigned char)*width*height*depth*nchannels);
  hipMalloc((void**)&d_img2_pixels, sizeof(unsigned char)*width*height*depth*nchannels);
  hipMalloc((void**)&d_ann, sizeof(unsigned char)*width*height*depth*nchannels);
  hipMalloc((void**)&d_annd, sizeof(unsigned char)*width*height*depth*4);
  hipMalloc((void**)&d_ann_buf, sizeof(unsigned char)*width*height*depth*nchannels);
  hipMalloc((void**)&d_annd_buf, sizeof(unsigned char)*width*height*depth*4);


  hipMemcpy(d_img1_pixels, img1_pixels, sizeof(unsigned char)*width*height*depth*nchannels, hipMemcpyHostToDevice);
  hipMemcpy(d_img2_pixels, img2_pixels, sizeof(unsigned char)*width*height*depth*nchannels, hipMemcpyHostToDevice);


  int aew = width - patch_width+1, aeh = height - patch_width + 1, aed = depth - patch_width + 1;       /* Effective width and height (possible upper left corners of patches). */
  int bew = width - patch_width+1, beh = height - patch_width + 1, bed = depth - patch_width + 1;
  printf ("%d %d %d %d %d %d\n", aew, aeh, aed, bew, beh, bed);
  for (int az = 0; az < aed; az++) {
    for (int ay = 0; ay < aeh; ay++) {
      for (int ax = 0; ax < aew; ax++){
        int bx = rand()%bew;
        int by = rand()%beh;
        int bz = rand()%bed;
        unsigned char* pixelOffset = get_RGB(ann, az, ay, ax, 3, h, w);
        unsigned char* bufOffsets = get_RGB(ann_buf, az, ay, ax, 3, h, w);
        storeXYasRGB(pixelOffset, bx, by, bz);
        storeXYasRGB(bufOffsets, bx, by, bz);
        unsigned char* distOffset = get_RGB(annd, az, ay, ax, 4, h, w);
        int distance = dist(img1_pixels, img2_pixels, ax, ay, az, bx, by, bz, patch_width, h, w);
        storeintasRGBA(distOffset, distance);
        unsigned char* distOffset_buf = get_RGB(annd_buf, az, ay, ax, 4, h, w);
        storeintasRGBA(distOffset_buf, distance);
      }
    }
  }

  printf("initialization done\n");

  hipMemcpy(d_ann, ann, sizeof(unsigned char)*width*height*depth*3, hipMemcpyHostToDevice);
    hipMemcpy(d_ann_buf, ann_buf, sizeof(unsigned char)*width*height*depth*3, hipMemcpyHostToDevice);
    hipMemcpy(d_annd, annd, sizeof(unsigned char)*width*height*depth*4, hipMemcpyHostToDevice);
    hipMemcpy(d_annd_buf, annd_buf, sizeof(unsigned char)*width*height*depth*4, hipMemcpyHostToDevice);

  // unsigned char* ann_to_use = NULL;
  // unsigned char* other = NULL;
  // unsigned char* annd_to_use = NULL;
  // unsigned char* annd_other = NULL;

    hipEvent_t start_k, stop_k;
    hipEventCreate(&start_k);
    hipEventCreate(&stop_k);

  hipEventRecord(start_k);

  for (int iter = 0; iter < patch_match_iters; iter++) {
    int zend = aed;
    int yend = aeh;
    int xend = aew;

    dim3 dimBlock(8, 8, 8);
    dim3 dimGrid(xend/8 + 1, yend/8 + 1, zend/8 + 1);
    // dim3 dimGrid(1, 1, 1);

    if (iter % 2 == 0){
      patchmatch<<< dimGrid, dimBlock >>>(d_img1_pixels, d_img2_pixels, d_ann, d_annd, d_ann_buf, d_annd_buf, height, width, depth, patch_width);
    }
    
    if (iter % 2 == 1) {
      patchmatch<<< dimGrid, dimBlock >>>(d_img1_pixels, d_img2_pixels, d_ann_buf, d_annd_buf, d_ann, d_annd, height, width, depth, patch_width);

    }
    // patchmatch<<<zend,yend,xend>>>(d_img1_pixels, d_img2_pixels, ann_to_use, annd_to_use, other, annd_other, height, width, depth, patch_width);
    hipDeviceSynchronize();
 }

 hipEventRecord(stop_k);



  cout << "Patch match done" << endl;

  hipEventSynchronize(stop_k);

  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start_k, stop_k);
  printf("\tTime measured for patchmatch is %.2f milliseconds.\n", milliseconds);

 

  unsigned char* output = (unsigned char*) malloc(sizeof(unsigned char)*width*height*depth*3);
  hipMemcpy(ann, d_ann, sizeof(unsigned char)*width*height*depth*3, hipMemcpyDeviceToHost);

  map_patches(img1_pixels, img2_pixels, ann, output);
  
  string output_folder = "./color/out/";
  string ann_folder = "./color/ann/";
  string annd_folder = "./color/annd/";
  
  for (int i = 0; i < d; i ++){
    int ann_start_idx = width *  height * i * nchannels;
    string ann_path= ann_folder+ to_string(i) +".png";
    stbi_write_png(ann_path.c_str(), width, height, 3, ann + ann_start_idx, width*3);
    
    int annd_start_idx = width * height * i * nchannels;
    string annd_path= annd_folder+ to_string(i) +".png";
    stbi_write_png(annd_path.c_str(), width, height, 4, annd + annd_start_idx, width*4);

    int output_start_idx = width * height * i * nchannels;
    string output_path= output_folder+ to_string(i) +".png";
    stbi_write_png(output_path.c_str(), width, height, 3, output + output_start_idx, width*3);
  }

  double error = compute_error(output, img1_pixels);
  long pixels = d*w*h;
  double per_pixel_error = error/pixels;
  cout << "The Error computed is " << error << endl;
  cout << "Per pixel error is "<<per_pixel_error << endl;
  
  return 0;
}
